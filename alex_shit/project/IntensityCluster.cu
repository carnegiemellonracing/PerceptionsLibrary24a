#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
// #include <vector>
// #include <cmath>
// #include <algorithm>
// #include <thrust/device_vector.h>
// #include <thrust/host_vector.h>
// #include <iostream>
// #include <unordered_map>


// // Was 10
// #define SECTION_RADIUS 10.0f
// #define SPLINE_SAMPLES 1000
// #define TRACK_WIDTH 3.0f



// struct Point {
//     float x, y, z, intensity;

//     __host__ __device__
//     Point(float x = 0, float y = 0, float z = 0, float intensity = 0)
//         : x(x), y(y), z(z), intensity(intensity) {}

//     __host__ __device__
//     bool operator==(const Point& other) const {
//         return (x == other.x) && (y == other.y) && (z == other.z) && (intensity == other.intensity);
//     }
// };

// struct SplineSegment {
//     float a, b, c, d; // Coefficients for the cubic spline
//     float x_start;    // Start of the segment
// };







// std::vector<std::vector<int>> NewRunDBSCAN(const std::vector<Point>& points, float eps, int min_samples) {
//     int num_points = points.size();
//     std::vector<std::vector<int>> clusters;

//     // Create visited and labels arrays
//     std::vector<bool> visited(num_points, false);
//     std::vector<int> labels(num_points, -1);  // -1 means noise
//     int cluster_id = 0;

//     // Helper lambda to calculate distance
//     auto distance = [](const Point& a, const Point& b) {
//         float dx = a.x - b.x;
//         float dy = a.y - b.y;
//         float dz = a.z - b.z;
//         return sqrtf(dx * dx + dy * dy + dz * dz);
//     };

//     // Helper function to find neighbors
//     auto get_neighbors = [&](int idx) {
//         std::vector<int> neighbors;
//         for (int i = 0; i < num_points; ++i) {
//             if (i != idx && distance(points[idx], points[i]) <= eps) {
//                 neighbors.push_back(i);
//             }
//         }
//         return neighbors;
//     };

//     // Main DBSCAN logic
//     for (int i = 0; i < num_points; ++i) {
//         if (visited[i]) continue;
//         visited[i] = true;

//         // Find neighbors
//         auto neighbors = get_neighbors(i);
//         if (neighbors.size() < min_samples) {
//             labels[i] = -1;  // Mark as noise
//             continue;
//         }

//         // Create a new cluster
//         clusters.emplace_back();
//         labels[i] = cluster_id;
//         clusters[cluster_id].push_back(i);

//         // Expand the cluster
//         std::vector<int> to_process = neighbors;
//         while (!to_process.empty()) {
//             int current = to_process.back();
//             to_process.pop_back();

//             if (!visited[current]) {
//                 visited[current] = true;

//                 // Get neighbors of the current point
//                 auto current_neighbors = get_neighbors(current);
//                 if (current_neighbors.size() >= min_samples) {
//                     to_process.insert(to_process.end(), current_neighbors.begin(), current_neighbors.end());
//                 }
//             }

//             if (labels[current] == -1 || labels[current] == -2) {
//                 labels[current] = cluster_id;
//                 clusters[cluster_id].push_back(current);
//             }
//         }

//         cluster_id++;
//     }

//     return clusters;
// }






// __global__ void computeMidlinePoints(const Point* sections, Point* midline, const int* section_sizes, int num_sections) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= num_sections) return;

//     const Point* section_start = sections + idx * section_sizes[idx];
//     int section_size = section_sizes[idx];

//     float sum_x = 0.0f, sum_y = 0.0f, sum_z = 0.0f;
//     for (int i = 0; i < section_size; ++i) {
//         sum_x += section_start[i].x;
//         sum_y += section_start[i].y;
//         sum_z += section_start[i].z;
//     }

//     midline[idx] = {sum_x / section_size, sum_y / section_size, sum_z / section_size, 0.0f};
// }




// // CUDA kernel to classify cones based on midline
// __global__ void classifyCones(const Point* cones, const Point* midline, int* labels, int num_cones, int num_midline_points) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= num_cones) return;

//     const Point& cone = cones[idx];
//     float min_dist = INFINITY;
//     int closest_segment = -1;

//     // Find the closest segment of the midline
//     for (int i = 0; i < num_midline_points - 1; ++i) {
//         float x1 = midline[i].x, y1 = midline[i].y;
//         float x2 = midline[i + 1].x, y2 = midline[i + 1].y;

//         // Calculate distance from the cone to the line segment
//         float dx = x2 - x1, dy = y2 - y1;
//         float t = ((cone.x - x1) * dx + (cone.y - y1) * dy) / (dx * dx + dy * dy);
//         t = fmaxf(0.0f, fminf(1.0f, t)); // Clamp t to [0, 1]
//         float proj_x = x1 + t * dx, proj_y = y1 + t * dy;
//         float dist = sqrtf((cone.x - proj_x) * (cone.x - proj_x) + (cone.y - proj_y) * (cone.y - proj_y));

//         if (dist < min_dist) {
//             min_dist = dist;
//             closest_segment = i;
//         }
//     }

//     // Classify based on relative position to the closest segment
//     const Point& p1 = midline[closest_segment];
//     const Point& p2 = midline[closest_segment + 1];
//     float normal_x = -(p2.y - p1.y), normal_y = p2.x - p1.x; // Perpendicular vector
//     float dx = cone.x - p1.x, dy = cone.y - p1.y;

//     labels[idx] = (dx * normal_x + dy * normal_y > 0) ? 1 : 0; // 1 for blue, 0 for yellow
// }




// __global__ void flattenSectionsKernel(const Point* sections, const int* section_offsets, const int* section_sizes, Point* flattened_sections, int total_sections) {
//     int section_idx = blockIdx.x; // Each block processes one section
//     if (section_idx >= total_sections) return;

//     int section_offset = section_offsets[section_idx];
//     int section_size = section_sizes[section_idx];

//     int thread_idx = threadIdx.x; // Thread ID within the section
//     if (thread_idx < section_size) {
//         flattened_sections[section_offset + thread_idx] = sections[section_offset + thread_idx];
//     }
// }








// std::vector<std::pair<Point, std::string>> ClusterCones(const std::vector<Point>& points) {


//     // Start the timer
//     auto start_time = std::chrono::high_resolution_clock::now();


//     // Step 1: Run DBSCAN to find clusters
//     auto clusters = NewRunDBSCAN(points, 7, 3);

//     // Step 2: Find the cluster closest to the origin
//     int best_cluster_idx = -1;
//     float min_dist_to_origin = INFINITY;

//     // Compute closest distance for each cluster in parallel
//     thrust::host_vector<float> min_distances(clusters.size());
//     thrust::host_vector<int> cluster_indices(clusters.size());

//     thrust::transform(
//         thrust::counting_iterator<int>(0),
//         thrust::counting_iterator<int>(clusters.size()),
//         min_distances.begin(),
//         [&](int cluster_idx) {
//             float min_dist = INFINITY;
//             for (int idx : clusters[cluster_idx]) {
//                 const Point& p = points[idx];
//                 float dist_to_origin = sqrtf(p.x * p.x + p.y * p.y);
//                 if (dist_to_origin < min_dist) {
//                     min_dist = dist_to_origin;
//                     cluster_indices[cluster_idx] = cluster_idx;
//                 }
//             }
//             return min_dist;
//         }
//     );

//     // Find the cluster closest to the origin
//     best_cluster_idx = thrust::min_element(min_distances.begin(), min_distances.end()) - min_distances.begin();

//     if (best_cluster_idx == -1) {
//         throw std::runtime_error("No valid cluster found closest to the origin.");
//     }

//     // Step 3: Re-run DBSCAN on the identified "track" cluster with smaller EPS
//     std::vector<Point> preprocessed_points;
//     for (int idx : clusters[best_cluster_idx]) {
//         preprocessed_points.push_back(points[idx]);
//     }

//     int num_points = preprocessed_points.size();

//     Point closest_positive_x, closest_negative_x;
//     float min_positive_x_dist = INFINITY, min_negative_x_dist = INFINITY;

//     // Step 4: Find the starting reference point
//     for (const auto& p : preprocessed_points) {
//         float dist = sqrtf(p.x * p.x + p.y * p.y);
//         if (p.x > 0 && dist < min_positive_x_dist) {
//             min_positive_x_dist = dist;
//             closest_positive_x = p;
//         } else if (p.x < 0 && dist < min_negative_x_dist) {
//             min_negative_x_dist = dist;
//             closest_negative_x = p;
//         }
//     }

//     if (min_positive_x_dist == INFINITY || min_negative_x_dist == INFINITY) {
//         std::cerr << "Fallback: Unable to find points in positive or negative x. Searching for fallback points.\n";

//         // Find the absolute closest point to the origin
//         Point closest1;
//         float min_dist1 = INFINITY;

//         for (const auto& p : preprocessed_points) {
//             float dist = sqrtf(p.x * p.x + p.y * p.y);
//             if (dist < min_dist1) {
//                 min_dist1 = dist;
//                 closest1 = p;
//             }
//         }

//         // Find the next closest point that is at least 1.5m away from closest1
//         Point closest2;
//         float min_dist2 = INFINITY;

//         for (const auto& p : preprocessed_points) {
//             float dist_to_origin = sqrtf(p.x * p.x + p.y * p.y);
//             float dist_to_closest1 = sqrtf(
//                 (p.x - closest1.x) * (p.x - closest1.x) +
//                 (p.y - closest1.y) * (p.y - closest1.y)
//             );

//             if (dist_to_origin < min_dist2 && dist_to_closest1 >= 1.5f) {
//                 min_dist2 = dist_to_origin;
//                 closest2 = p;
//             }
//         }

//         if (min_dist2 == INFINITY) {
//             throw std::runtime_error("Fallback method failed: No second point found that is at least 1.5m away from the closest point.");
//         }

//         // Set fallback points as closest_positive_x and closest_negative_x
//         closest_positive_x = closest1;
//         closest_negative_x = closest2;

//         std::cerr << "Fallback successful: Closest points found are spaced more than 1.5m apart.\n";
//     }

//     Point current_ref = {
//         (closest_positive_x.x + closest_negative_x.x) / 2.0f,
//         (closest_positive_x.y + closest_negative_x.y) / 2.0f,
//         (closest_positive_x.z + closest_negative_x.z) / 2.0f,
//         0.0f
//     };



//     // Step 5: Section points and record the start and end points
//     std::vector<std::vector<Point>> sections;
//     std::vector<Point> section_start_points;
//     std::vector<Point> remaining_points = preprocessed_points;

//     while (!remaining_points.empty()) {
//         std::vector<Point> current_section;
//         auto it = remaining_points.begin();
//         while (it != remaining_points.end()) {
//             float dx = it->x - current_ref.x, dy = it->y - current_ref.y;
//             float dist_to_current_section = sqrtf(dx * dx + dy * dy);

//             // Check overlap with previous section centers
//             float dist_to_previous_section = INFINITY;
//             if (!sections.empty()) {
//                 const Point& previous_center = section_start_points.back();
//                 float dx_prev = it->x - previous_center.x, dy_prev = it->y - previous_center.y;
//                 dist_to_previous_section = sqrtf(dx_prev * dx_prev + dy_prev * dy_prev);
//             }

//             // Assign point only if it's closer to the current section center
//             if (dist_to_current_section <= SECTION_RADIUS &&
//                 dist_to_current_section < dist_to_previous_section) {
//                 current_section.push_back(*it);
//                 it = remaining_points.erase(it);
//             } else {
//                 ++it;
//             }
//         }

//         if (current_section.empty()) break;

//         sections.push_back(current_section);
//         section_start_points.push_back(current_ref);

//         // Update the reference point
//         if (!current_section.empty()) {
//             float max_dist1 = 0.0f, max_dist2 = 0.0f;
//             Point furthest1 = current_section[0], furthest2 = current_section[0];
//             for (const auto& point : current_section) {
//                 float dist = sqrtf((point.x - current_ref.x) * (point.x - current_ref.x) +
//                                 (point.y - current_ref.y) * (point.y - current_ref.y));
//                 if (dist > max_dist1) {
//                     max_dist2 = max_dist1;
//                     furthest2 = furthest1;
//                     max_dist1 = dist;
//                     furthest1 = point;
//                 } else if (dist > max_dist2) {
//                     max_dist2 = dist;
//                     furthest2 = point;
//                 }
//             }
//             current_ref = {
//                 (furthest1.x + furthest2.x) / 2.0f,
//                 (furthest1.y + furthest2.y) / 2.0f,
//                 (furthest1.z + furthest2.z) / 2.0f,
//                 0.0f
//             };
//         }
//     }

//     // Flatten sections into a single array for midline computation
//     std::vector<Point> flattened_sections;
//     std::vector<int> section_sizes;
//     for (const auto& section : sections) {
//         flattened_sections.insert(flattened_sections.end(), section.begin(), section.end());
//         section_sizes.push_back(section.size());
//     }

//     // Compute midline points
//     thrust::device_vector<Point> d_sections(flattened_sections);
//     thrust::device_vector<int> d_section_sizes(section_sizes);
//     thrust::device_vector<Point> d_midline(sections.size());

//     computeMidlinePoints<<<(sections.size() + 255) / 256, 256>>>(
//         thrust::raw_pointer_cast(d_sections.data()),
//         thrust::raw_pointer_cast(d_midline.data()),
//         thrust::raw_pointer_cast(d_section_sizes.data()),
//         sections.size());
//     hipDeviceSynchronize();

//     thrust::host_vector<Point> midline = d_midline;

//     // Add the first and last points to the midline
//     midline.insert(midline.begin(), section_start_points.front()); // Add start point
//     midline.push_back(current_ref); // Add end point

//     // Classify cones based on the updated midline
//     thrust::device_vector<int> d_labels(num_points);
//     thrust::device_vector<Point> d_points(preprocessed_points);
//     thrust::device_vector<Point> d_updated_midline(midline);
//     classifyCones<<<(num_points + 255) / 256, 256>>>(
//         thrust::raw_pointer_cast(d_points.data()),
//         thrust::raw_pointer_cast(d_updated_midline.data()),
//         thrust::raw_pointer_cast(d_labels.data()),
//         num_points, midline.size());
//     hipDeviceSynchronize();

//     thrust::host_vector<int> labels = d_labels;

//     // Stop the timer
//     auto end_time = std::chrono::high_resolution_clock::now();
//     auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count();
//     std::cout << "ClusterCones execution time: " << duration << " µs" << std::endl;

//     // Create output with midline points labeled as "green" and section start points labeled as "section"
//     std::vector<std::pair<Point, std::string>> classified_cones;
//     for (int i = 0; i < num_points; ++i) {
//         classified_cones.emplace_back(preprocessed_points[i], (labels[i] == 1) ? "blue" : "yellow");
//     }
//     for (const auto& p : midline) {
//         classified_cones.emplace_back(p, "green");
//     }
//     for (const auto& p : section_start_points) {
//         classified_cones.emplace_back(p, "section");
//     }

//     return classified_cones;
// }













#include <vector>
#include <cmath>
#include <string>
#include <algorithm>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "Point.hpp"
#include <algorithm>
#include <stdexcept>

// Global constants
const float EPS = 5.5f;       // Large `eps` for clustering
const int MIN_SAMPLES = 3;    // Minimum points for a valid cluster
const float TRACK_WIDTH = 3.0f; // Approximate track width for labeling







std::vector<std::vector<int>> NewRunDBSCAN(const std::vector<Point>& points, float eps, int min_samples) {
    int num_points = points.size();
    std::vector<std::vector<int>> clusters;

    // Create visited and labels arrays
    std::vector<bool> visited(num_points, false);
    std::vector<int> labels(num_points, -1);  // -1 means noise
    int cluster_id = 0;

    // Helper lambda to calculate distance
    auto distance = [](const Point& a, const Point& b) {
        float dx = a.x - b.x;
        float dy = a.y - b.y;
        float dz = a.z - b.z;
        return sqrtf(dx * dx + dy * dy + dz * dz);
    };

    // Helper function to find neighbors
    auto get_neighbors = [&](int idx) {
        std::vector<int> neighbors;
        for (int i = 0; i < num_points; ++i) {
            if (i != idx && distance(points[idx], points[i]) <= eps) {
                neighbors.push_back(i);
            }
        }
        return neighbors;
    };

    // Main DBSCAN logic
    for (int i = 0; i < num_points; ++i) {
        if (visited[i]) continue;
        visited[i] = true;

        // Find neighbors
        auto neighbors = get_neighbors(i);
        if (neighbors.size() < min_samples) {
            labels[i] = -1;  // Mark as noise
            continue;
        }

        // Create a new cluster
        clusters.emplace_back();
        labels[i] = cluster_id;
        clusters[cluster_id].push_back(i);

        // Expand the cluster
        std::vector<int> to_process = neighbors;
        while (!to_process.empty()) {
            int current = to_process.back();
            to_process.pop_back();

            if (!visited[current]) {
                visited[current] = true;

                // Get neighbors of the current point
                auto current_neighbors = get_neighbors(current);
                if (current_neighbors.size() >= min_samples) {
                    to_process.insert(to_process.end(), current_neighbors.begin(), current_neighbors.end());
                }
            }

            if (labels[current] == -1 || labels[current] == -2) {
                labels[current] = cluster_id;
                clusters[cluster_id].push_back(current);
            }
        }

        cluster_id++;
    }

    return clusters;
}



std::vector<std::pair<Point, std::string>> labelConesWithDBSCAN(
    const std::vector<Point>& points,
    const std::vector<int>& cluster,
    float eps_for_coloring,
    int min_samples_for_coloring) {

    // Extract the points in the "track" cluster
    std::vector<Point> track_points;
    for (int idx : cluster) {
        track_points.push_back(points[idx]);
    }

    // Run DBSCAN again with smaller EPS
    auto sub_clusters = NewRunDBSCAN(track_points, eps_for_coloring, min_samples_for_coloring);

    // Debug: Print number of sub-clusters found
    if (sub_clusters.size() < 2) {
        std::cerr << "Error: Not enough sub-clusters found for labeling as left/right bounds." << std::endl;
        return {};
    }

    if (sub_clusters.size() < 2) {
        throw std::runtime_error("Not enough sub-clusters found for labeling as left/right bounds.");
    }

    // If more than two clusters, find the two closest to the origin
    if (sub_clusters.size() > 2) {
        std::vector<std::pair<float, int>> cluster_distances; // (distance_to_origin, cluster_index)

        for (size_t i = 0; i < sub_clusters.size(); ++i) {
            float min_dist_to_origin = INFINITY;
            for (int idx : sub_clusters[i]) {
                const Point& p = track_points[idx];
                float dist_to_origin = sqrtf(p.x * p.x + p.y * p.y);
                min_dist_to_origin = std::min(min_dist_to_origin, dist_to_origin);
            }
            cluster_distances.emplace_back(min_dist_to_origin, i);
        }

        // Sort clusters by distance to origin
        std::sort(cluster_distances.begin(), cluster_distances.end());

        // Select the two closest clusters
        sub_clusters = {sub_clusters[cluster_distances[0].second], sub_clusters[cluster_distances[1].second]};
    }

    // Assign labels to the two clusters
    std::vector<std::pair<Point, std::string>> labeled_cones;

    // Ensure consistent ordering: label the cluster with smaller mean x-coordinate as "yellow"
    float mean_x_cluster_0 = 0, mean_x_cluster_1 = 0;
    for (int idx : sub_clusters[0]) {
        mean_x_cluster_0 += track_points[idx].x;
    }
    for (int idx : sub_clusters[1]) {
        mean_x_cluster_1 += track_points[idx].x;
    }
    mean_x_cluster_0 /= sub_clusters[0].size();
    mean_x_cluster_1 /= sub_clusters[1].size();

    bool cluster_0_is_yellow = mean_x_cluster_0 < mean_x_cluster_1;

    for (size_t i = 0; i < sub_clusters.size(); ++i) {
        std::string label = (cluster_0_is_yellow == (i == 0)) ? "yellow" : "blue";
        for (int idx : sub_clusters[i]) {
            labeled_cones.emplace_back(track_points[idx], label);
        }
    }

    return labeled_cones;
}



// Main pipeline
std::vector<std::pair<Point, std::string>> ClusterCones(const std::vector<Point>& points) {

    // Start timing
    auto start = std::chrono::high_resolution_clock::now();

    // Step 1: Run DBSCAN with large EPS to identify the track cluster
    auto clusters = NewRunDBSCAN(points, EPS, MIN_SAMPLES);

    // Step 2: Find the cluster closest to the origin
    int best_cluster_idx = -1;
    float min_dist_to_origin = INFINITY;

    for (size_t i = 0; i < clusters.size(); ++i) {
        for (size_t j = 0; j < clusters[i].size(); ++j) {
            int idx = clusters[i][j];
            const Point& p = points[idx];
            float dist_to_origin = sqrtf(p.x * p.x + p.y * p.y);
            if (dist_to_origin < min_dist_to_origin) {
                min_dist_to_origin = dist_to_origin;
                best_cluster_idx = i;
            }
        }
    }

    if (best_cluster_idx == -1) {
        throw std::runtime_error("No valid cluster found closest to the origin.");
    }

    // Step 3: Re-run DBSCAN on the identified "track" cluster with smaller EPS
    const auto& track_cluster = clusters[best_cluster_idx];
    float small_eps = 2.8f; // Smaller EPS for sub-clustering
    int min_samples_for_coloring = 2; // Minimum samples for sub-clustering

    auto labeled_cones = labelConesWithDBSCAN(points, track_cluster, small_eps, min_samples_for_coloring);

    // End timing
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed = end - start;

    // Print execution time
    std::cout << "ClusterCones execution time: " << elapsed.count() << " ms" << std::endl;

    return labeled_cones;
}