#include "hip/hip_runtime.h"
// #include <hip/hip_runtime.h>
// #include <vector>
// #include <cmath>
// #include <algorithm>
// #include <thrust/device_vector.h>
// #include <thrust/host_vector.h>
// #include <iostream>
// #include <unordered_map>
// #include "Point.hpp"


// // Was 10
// #define SECTION_RADIUS 10.0f
// #define SPLINE_SAMPLES 1000
// #define TRACK_WIDTH 3.0f



// struct SplineSegment {
//     float a, b, c, d; // Coefficients for the cubic spline
//     float x_start;    // Start of the segment
// };



// std::vector<std::vector<int>> NewRunDBSCAN(const std::vector<Point>& points, float eps, int min_samples) {
//     int num_points = points.size();
//     std::vector<std::vector<int>> clusters;

//     // Create visited and labels arrays
//     std::vector<bool> visited(num_points, false);
//     std::vector<int> labels(num_points, -1);  // -1 means noise
//     int cluster_id = 0;

//     // Helper lambda to calculate distance
//     auto distance = [](const Point& a, const Point& b) {
//         float dx = a.x - b.x;
//         float dy = a.y - b.y;
//         float dz = a.z - b.z;
//         return sqrtf(dx * dx + dy * dy + dz * dz);
//     };

//     // Helper function to find neighbors
//     auto get_neighbors = [&](int idx) {
//         std::vector<int> neighbors;
//         for (int i = 0; i < num_points; ++i) {
//             if (i != idx && distance(points[idx], points[i]) <= eps) {
//                 neighbors.push_back(i);
//             }
//         }
//         return neighbors;
//     };

//     // Main DBSCAN logic
//     for (int i = 0; i < num_points; ++i) {
//         if (visited[i]) continue;
//         visited[i] = true;

//         // Find neighbors
//         auto neighbors = get_neighbors(i);
//         if (neighbors.size() < min_samples) {
//             labels[i] = -1;  // Mark as noise
//             continue;
//         }

//         // Create a new cluster
//         clusters.emplace_back();
//         labels[i] = cluster_id;
//         clusters[cluster_id].push_back(i);

//         // Expand the cluster
//         std::vector<int> to_process = neighbors;
//         while (!to_process.empty()) {
//             int current = to_process.back();
//             to_process.pop_back();

//             if (!visited[current]) {
//                 visited[current] = true;

//                 // Get neighbors of the current point
//                 auto current_neighbors = get_neighbors(current);
//                 if (current_neighbors.size() >= min_samples) {
//                     to_process.insert(to_process.end(), current_neighbors.begin(), current_neighbors.end());
//                 }
//             }

//             if (labels[current] == -1 || labels[current] == -2) {
//                 labels[current] = cluster_id;
//                 clusters[cluster_id].push_back(current);
//             }
//         }

//         cluster_id++;
//     }

//     return clusters;
// }


// __global__ void computeMidlinePoints(const Point* sections, Point* midline, const int* section_sizes, int num_sections) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= num_sections) return;

//     const Point* section_start = sections + idx * section_sizes[idx];
//     int section_size = section_sizes[idx];

//     float sum_x = 0.0f, sum_y = 0.0f, sum_z = 0.0f;
//     for (int i = 0; i < section_size; ++i) {
//         sum_x += section_start[i].x;
//         sum_y += section_start[i].y;
//         sum_z += section_start[i].z;
//     }

//     midline[idx] = {sum_x / section_size, sum_y / section_size, sum_z / section_size, 0.0f};
// }




// // CUDA kernel to classify cones based on midline
// __global__ void classifyCones(const Point* cones, const Point* midline, int* labels, int num_cones, int num_midline_points) {
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     if (idx >= num_cones) return;

//     const Point& cone = cones[idx];
//     float min_dist = INFINITY;
//     int closest_segment = -1;

//     // Find the closest segment of the midline
//     for (int i = 0; i < num_midline_points - 1; ++i) {
//         float x1 = midline[i].x, y1 = midline[i].y;
//         float x2 = midline[i + 1].x, y2 = midline[i + 1].y;

//         // Calculate distance from the cone to the line segment
//         float dx = x2 - x1, dy = y2 - y1;
//         float t = ((cone.x - x1) * dx + (cone.y - y1) * dy) / (dx * dx + dy * dy);
//         t = fmaxf(0.0f, fminf(1.0f, t)); // Clamp t to [0, 1]
//         float proj_x = x1 + t * dx, proj_y = y1 + t * dy;
//         float dist = sqrtf((cone.x - proj_x) * (cone.x - proj_x) + (cone.y - proj_y) * (cone.y - proj_y));

//         if (dist < min_dist) {
//             min_dist = dist;
//             closest_segment = i;
//         }
//     }

//     // Classify based on relative position to the closest segment
//     const Point& p1 = midline[closest_segment];
//     const Point& p2 = midline[closest_segment + 1];
//     float normal_x = -(p2.y - p1.y), normal_y = p2.x - p1.x; // Perpendicular vector
//     float dx = cone.x - p1.x, dy = cone.y - p1.y;

//     labels[idx] = (dx * normal_x + dy * normal_y > 0) ? 1 : 0; // 1 for blue, 0 for yellow
// }





// std::vector<std::pair<Point, std::string>> ColorCones(const std::vector<Point>& points) {


//     // Start the timer
//     auto start_time = std::chrono::high_resolution_clock::now();


//     // Step 1: Run DBSCAN to find clusters
//     auto clusters = NewRunDBSCAN(points, 7, 3);

//     // Step 2: Find the cluster closest to the origin
//     int best_cluster_idx = -1;

//     // Compute closest distance for each cluster in parallel
//     thrust::host_vector<float> min_distances(clusters.size());
//     thrust::host_vector<int> cluster_indices(clusters.size());

//     thrust::transform(
//         thrust::counting_iterator<int>(0),
//         thrust::counting_iterator<int>(clusters.size()),
//         min_distances.begin(),
//         [&](int cluster_idx) {
//             float min_dist = INFINITY;
//             for (int idx : clusters[cluster_idx]) {
//                 const Point& p = points[idx];
//                 float dist_to_origin = sqrtf(p.x * p.x + p.y * p.y);
//                 if (dist_to_origin < min_dist) {
//                     min_dist = dist_to_origin;
//                     cluster_indices[cluster_idx] = cluster_idx;
//                 }
//             }
//             return min_dist;
//         }
//     );

//     // Find the cluster closest to the origin
//     best_cluster_idx = thrust::min_element(min_distances.begin(), min_distances.end()) - min_distances.begin();

//     if (best_cluster_idx == -1) {
//         throw std::runtime_error("No valid cluster found closest to the origin.");
//     }

//     // Step 3: make preprocessed points
//     std::vector<Point> preprocessed_points;
//     for (int idx : clusters[best_cluster_idx]) {
//         preprocessed_points.push_back(points[idx]);
//     }

//     int num_points = preprocessed_points.size();

//     Point closest_positive_x, closest_negative_x;
//     float min_positive_x_dist = INFINITY, min_negative_x_dist = INFINITY;

//     // Step 4: Find the starting reference point
//     for (const auto& p : preprocessed_points) {
//         float dist = sqrtf(p.x * p.x + p.y * p.y);
//         if (p.x > 0 && dist < min_positive_x_dist) {
//             min_positive_x_dist = dist;
//             closest_positive_x = p;
//         } else if (p.x < 0 && dist < min_negative_x_dist) {
//             min_negative_x_dist = dist;
//             closest_negative_x = p;
//         }
//     }

//     if (min_positive_x_dist == INFINITY || min_negative_x_dist == INFINITY) {
//         std::cerr << "Fallback: Unable to find points in positive or negative x. Searching for fallback points.\n";

//         // Find the absolute closest point to the origin
//         Point closest1;
//         float min_dist1 = INFINITY;

//         for (const auto& p : preprocessed_points) {
//             float dist = sqrtf(p.x * p.x + p.y * p.y);
//             if (dist < min_dist1) {
//                 min_dist1 = dist;
//                 closest1 = p;
//             }
//         }

//         // Find the next closest point that is at least 1.5m away from closest1
//         Point closest2;
//         float min_dist2 = INFINITY;

//         for (const auto& p : preprocessed_points) {
//             float dist_to_origin = sqrtf(p.x * p.x + p.y * p.y);
//             float dist_to_closest1 = sqrtf(
//                 (p.x - closest1.x) * (p.x - closest1.x) +
//                 (p.y - closest1.y) * (p.y - closest1.y)
//             );

//             if (dist_to_origin < min_dist2 && dist_to_closest1 >= 1.5f) {
//                 min_dist2 = dist_to_origin;
//                 closest2 = p;
//             }
//         }

//         if (min_dist2 == INFINITY) {
//             throw std::runtime_error("Fallback method failed: No second point found that is at least 1.5m away from the closest point.");
//         }

//         // Set fallback points as closest_positive_x and closest_negative_x
//         closest_positive_x = closest1;
//         closest_negative_x = closest2;

//         std::cerr << "Fallback successful: Closest points found are spaced more than 1.5m apart.\n";
//     }

//     Point current_ref = {
//         (closest_positive_x.x + closest_negative_x.x) / 2.0f,
//         (closest_positive_x.y + closest_negative_x.y) / 2.0f,
//         (closest_positive_x.z + closest_negative_x.z) / 2.0f,
//         0.0f
//     };



//     // Step 5: Section points and record the start and end points
//     std::vector<std::vector<Point>> sections;
//     std::vector<Point> section_start_points;
//     std::vector<Point> remaining_points = preprocessed_points;

//     while (!remaining_points.empty()) {
//         std::vector<Point> current_section;
//         auto it = remaining_points.begin();
//         while (it != remaining_points.end()) {
//             float dx = it->x - current_ref.x, dy = it->y - current_ref.y;
//             float dist_to_current_section = sqrtf(dx * dx + dy * dy);

//             // Check overlap with previous section centers
//             float dist_to_previous_section = INFINITY;
//             if (!sections.empty()) {
//                 const Point& previous_center = section_start_points.back();
//                 float dx_prev = it->x - previous_center.x, dy_prev = it->y - previous_center.y;
//                 dist_to_previous_section = sqrtf(dx_prev * dx_prev + dy_prev * dy_prev);
//             }

//             // Assign point only if it's closer to the current section center
//             if (dist_to_current_section <= SECTION_RADIUS &&
//                 dist_to_current_section < dist_to_previous_section) {
//                 current_section.push_back(*it);
//                 it = remaining_points.erase(it);
//             } else {
//                 ++it;
//             }
//         }

//         if (current_section.empty()) break;

//         sections.push_back(current_section);
//         section_start_points.push_back(current_ref);

//         // Update the reference point
//         if (!current_section.empty()) {
//             float max_dist1 = 0.0f, max_dist2 = 0.0f;
//             Point furthest1 = current_section[0], furthest2 = current_section[0];
//             for (const auto& point : current_section) {
//                 float dist = sqrtf((point.x - current_ref.x) * (point.x - current_ref.x) +
//                                 (point.y - current_ref.y) * (point.y - current_ref.y));
//                 if (dist > max_dist1) {
//                     max_dist2 = max_dist1;
//                     furthest2 = furthest1;
//                     max_dist1 = dist;
//                     furthest1 = point;
//                 } else if (dist > max_dist2) {
//                     max_dist2 = dist;
//                     furthest2 = point;
//                 }
//             }
//             current_ref = {
//                 (furthest1.x + furthest2.x) / 2.0f,
//                 (furthest1.y + furthest2.y) / 2.0f,
//                 (furthest1.z + furthest2.z) / 2.0f,
//                 0.0f
//             };
//         }
//     }

//     // Flatten sections into a single array for midline computation
//     std::vector<Point> flattened_sections;
//     std::vector<int> section_sizes;
//     for (const auto& section : sections) {
//         flattened_sections.insert(flattened_sections.end(), section.begin(), section.end());
//         section_sizes.push_back(section.size());
//     }

//     // Compute midline points
//     thrust::device_vector<Point> d_sections(flattened_sections);
//     thrust::device_vector<int> d_section_sizes(section_sizes);
//     thrust::device_vector<Point> d_midline(sections.size());

//     computeMidlinePoints<<<(sections.size() + 255) / 256, 256>>>(
//         thrust::raw_pointer_cast(d_sections.data()),
//         thrust::raw_pointer_cast(d_midline.data()),
//         thrust::raw_pointer_cast(d_section_sizes.data()),
//         sections.size());
//     hipDeviceSynchronize();

//     thrust::host_vector<Point> midline = d_midline;

//     // Add the first and last points to the midline
//     midline.insert(midline.begin(), section_start_points.front()); // Add start point
//     midline.push_back(current_ref); // Add end point

//     // Classify cones based on the updated midline
//     thrust::device_vector<int> d_labels(num_points);
//     thrust::device_vector<Point> d_points(preprocessed_points);
//     thrust::device_vector<Point> d_updated_midline(midline);
//     classifyCones<<<(num_points + 255) / 256, 256>>>(
//         thrust::raw_pointer_cast(d_points.data()),
//         thrust::raw_pointer_cast(d_updated_midline.data()),
//         thrust::raw_pointer_cast(d_labels.data()),
//         num_points, midline.size());
//     hipDeviceSynchronize();

//     thrust::host_vector<int> labels = d_labels;

//     // Stop the timer
//     auto end_time = std::chrono::high_resolution_clock::now();
//     auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count();
//     std::cout << "ColorCones execution time: " << duration << " µs" << std::endl;

//     // Create output with midline points labeled as "green" and section start points labeled as "section"
//     std::vector<std::pair<Point, std::string>> classified_cones;
//     for (int i = 0; i < num_points; ++i) {
//         classified_cones.emplace_back(preprocessed_points[i], (labels[i] == 1) ? "blue" : "yellow");
//     }
//     for (const auto& p : midline) {
//         classified_cones.emplace_back(p, "green");
//     }
//     for (const auto& p : section_start_points) {
//         classified_cones.emplace_back(p, "section");
//     }

//     return classified_cones;
// }






#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>
#include <cmath>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "Point.hpp"
#include "DBscan.cuh"

// Helper function to calculate the number of grid blocks
inline dim3 calculateGridSize(int num_points, int block_size) {
    return dim3((num_points + block_size - 1) / block_size);
}

// Parallelized DBSCAN implementation
std::vector<int> runParallelDBSCAN(const std::vector<Point>& points, float eps, int min_samples) {
    int num_points = points.size();

    // Allocate device memory
    thrust::device_vector<Point> d_points(points.begin(), points.end());
    thrust::device_vector<int> d_parent(num_points);
    thrust::device_vector<int> d_labels(num_points, -1); // -1 indicates noise
    thrust::device_vector<int> d_cluster_sizes(num_points, 0);
    thrust::device_vector<int> d_is_core(num_points, 0); // 1 if point is core

    // Step 1: Initialize clusters
    dim3 block(256);
    dim3 grid = calculateGridSize(num_points, block.x);
    initializeClusters<<<grid, block>>>(thrust::raw_pointer_cast(d_parent.data()), num_points);
    hipDeviceSynchronize();

    // Step 2: Find and union clusters
    findAndUnionClusters<<<grid, block>>>(
        thrust::raw_pointer_cast(d_points.data()),
        thrust::raw_pointer_cast(d_parent.data()),
        num_points, eps);
    hipDeviceSynchronize();

    // Step 3: Flatten clusters for path compression
    flattenClusters<<<grid, block>>>(thrust::raw_pointer_cast(d_parent.data()), num_points);
    hipDeviceSynchronize();

    // Step 4: Assign cluster IDs
    thrust::host_vector<int> h_parent = d_parent; // Copy parents back to host
    std::unordered_map<int, int> cluster_map;
    int next_cluster_id = 0;

    for (int i = 0; i < num_points; ++i) {
        int root = h_parent[i];
        if (cluster_map.find(root) == cluster_map.end()) {
            cluster_map[root] = next_cluster_id++;
        }
        h_parent[i] = cluster_map[root];
    }

    // Copy back updated cluster IDs to the device
    thrust::device_vector<int> d_cluster_ids(h_parent.begin(), h_parent.end());

    // Copy cluster IDs to host for returning
    thrust::host_vector<int> h_cluster_ids = d_cluster_ids;

    return std::vector<int>(h_cluster_ids.begin(), h_cluster_ids.end());
}

// Parallelized main pipeline
std::vector<std::pair<Point, std::string>> ColorCones(const std::vector<Point>& points) {
    auto start = std::chrono::high_resolution_clock::now();
    
    int num_points = points.size();

    // Step 1: Run DBSCAN to identify clusters
    auto cluster_ids = runParallelDBSCAN(points, 5.5f, 3);

    // Step 2: Find the cluster closest to the origin
    std::unordered_map<int, Point> cluster_centroids;
    std::unordered_map<int, int> cluster_sizes;
    int best_cluster_id = -1;
    float min_dist_to_origin = INFINITY;

    for (int i = 0; i < num_points; ++i) {
        int cluster_id = cluster_ids[i];
        if (cluster_id >= 0) {
            const Point& p = points[i];
            float dist_to_origin = sqrtf(p.x * p.x + p.y * p.y);

            // Update cluster centroid calculations
            cluster_centroids[cluster_id].x += p.x;
            cluster_centroids[cluster_id].y += p.y;
            cluster_sizes[cluster_id]++;

            // Track the closest cluster to the origin
            if (dist_to_origin < min_dist_to_origin) {
                min_dist_to_origin = dist_to_origin;
                best_cluster_id = cluster_id;
            }
        }
    }

    if (best_cluster_id == -1) {
        throw std::runtime_error("No valid cluster found closest to the origin.");
    }

    // Compute final centroids
    for (auto& [id, centroid] : cluster_centroids) {
        int size = cluster_sizes[id];
        centroid.x /= size;
        centroid.y /= size;
    }

    // Extract points belonging to the best cluster
    std::vector<Point> track_points;
    for (int i = 0; i < num_points; ++i) {
        if (cluster_ids[i] == best_cluster_id) {
            track_points.push_back(points[i]);
        }
    }

    // Step 3: Sub-cluster the track points with smaller EPS
    float small_eps = 2.8f;
    int min_samples_for_coloring = 2;
    auto sub_cluster_ids = runParallelDBSCAN(track_points, small_eps, min_samples_for_coloring);

    // Step 4: Assign labels based on sub-cluster membership
    std::vector<std::pair<Point, std::string>> labeled_cones;
    std::unordered_map<int, int> sub_cluster_sizes;

    for (int id : sub_cluster_ids) {
        if (id >= 0) {
            sub_cluster_sizes[id]++;
        }
    }

    if (sub_cluster_sizes.size() < 2) {
        throw std::runtime_error("Not enough sub-clusters found for left/right labeling.");
    }

    // Find two largest sub-clusters
    std::vector<std::pair<int, int>> sorted_clusters(sub_cluster_sizes.begin(), sub_cluster_sizes.end());
    std::sort(sorted_clusters.begin(), sorted_clusters.end(), [](const auto& a, const auto& b) {
        return b.second < a.second; // Sort descending by size
    });

    int left_cluster_id = sorted_clusters[0].first;
    int right_cluster_id = sorted_clusters[1].first;

    for (int i = 0; i < track_points.size(); ++i) {
        int sub_cluster_id = sub_cluster_ids[i];
        if (sub_cluster_id == left_cluster_id) {
            labeled_cones.emplace_back(track_points[i], "blue");
        } else if (sub_cluster_id == right_cluster_id) {
            labeled_cones.emplace_back(track_points[i], "yellow");
        }
    }


    // Stop the timer
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed_time = end - start;

    // Print the timing results
    std::cout << "Clustered coloring execution time: " << elapsed_time.count() << " ms" << std::endl;

    return labeled_cones;
}